
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

__global__
void saxpy(int n, float a, float * x, float * y)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
//  printf("i = %d \n", i);
  if (i < n){
    y[i] = a*x[i] + y[i];
  }
}


int main(void) {
  int N = 1<<20;

  float *x; // = new float[N*sizeof(float)];
  float *y; // = new float[N*sizeof(float)];
  
  x = (float*)malloc(N*sizeof(float));
  y = (float*)malloc(N*sizeof(float));
  
  float *d_x, *d_y;

  // initialize x and y arrays on the host
  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  hipMalloc(&d_x, N*sizeof(float));
  hipMalloc(&d_y, N*sizeof(float));
  
  hipMemcpy(d_x, x, N*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_y, y, N*sizeof(float), hipMemcpyHostToDevice);

  saxpy<<<4096,256>>>(N, 2.0f, d_x, d_y);
  
  // Wait for GPU to finish before accessing on host
  hipMemcpy(y, d_y, N*sizeof(float), hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
  
  // Check for errors (all values should be 3.0f)
  float maxError = 0.0f;
  for (int i = 0; i < N; i++)
    maxError = fmax(maxError, fabs(y[i]-4.0f));
  std::cout << "Max error: " << maxError << std::endl;

	

  // Free memory
  delete x;
  delete y;
  hipFree(d_x);
  hipFree(d_y);

  return 0;
}
